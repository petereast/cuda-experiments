#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

void getInfo( void )
{
    hipDeviceProp_t p;

    int capability = p.major;

    if(capability)
    {
        hipGetDeviceProperties( &p, 0);
        printf(" -- Information & Properties about CUDA device 0 -- \n");
        printf("\tCompute Capability: %i.%i\n", p.major, p.minor);
        printf("\tDevice Name: %s\n", p.name);
        printf("\tClock Rate: %d\n", p.clockRate);
        printf("\tGlobal Memory: %dMiB\n", p.totalGlobalMem/(1024*1024));
        printf(" -- End of Information -- \n");
    }else
    {
        printf(" -- Warning: No CUDA Device Detected :'( -- \n");
    }


}

// Let's do a basic CAESAR shift cipher, implemented in CUDA
__global__ void caeasrCipher(char *key, char *text, int tlength, int klength);


int main( void ){

    getInfo();

    char *key = "Hellfo";
    char *text = "The Quick Brown Fox Jumped over The Lazy Dawg";

    char *dev_key, *dev_text;

    hipMalloc( (void**)&dev_key, strlen(key)*sizeof(char));
    hipMalloc( (void**)&dev_text, strlen(text)*sizeof(char));

    hipMemcpy( dev_key, key, strlen(key)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( dev_text, text, (strlen(text)+1)*sizeof(char), hipMemcpyHostToDevice);
    printf("Key: %s(%d)\nText: '%s'(%d)\n", key, strlen(key), text, strlen(text));
    caeasrCipher<<<1024, 1>>>(dev_key, dev_text, strlen(text), strlen(key));

    hipMemcpy(text, dev_text, (strlen(text)+1)*sizeof(char), hipMemcpyDeviceToHost);

    printf("Output: %s\n", text);

    hipFree(dev_text);
    hipFree(dev_key);


    return 0;
}

__global__ void caeasrCipher(char *key, char *text, int tlength, int klength)
{
    int tid = blockIdx.x;
    if (tid < tlength)
    {
        printf("%c - %d\n",text[tid], tid);
        (char*)text[tid] = ((int)text[tid] + (int)key[tid % klength])%127;
        printf("%c - %d\n",text[tid], tid);
    }

}
