#include "hip/hip_runtime.h"
// This turned out to be a little more complex than I had first thought, maybe I should try a different project.
//


#include <iostream>
#include <stdio.h>


void getInfo( void );
bool getDevicePresent( void );

// Let's do a basic CAESAR shift cipher, implemented in CUDA
__global__ void caesarCipher(char *key, char *text, int tlength, int klength);
__global__ void unCaesarCipher(char *key, char *text, int tlength, int klength);


int main( void ){

    getInfo();

    if(!getDevicePresent())
    {
        return -1;
    }

    char *key = "fsdbikjb";
    char *text = "The Quick Brown Fox Jumped over The Lazy Dawg";

    char *dev_key, *dev_text;

    int textSize =(strlen(text) * sizeof(char))+1;
    int keySize = (strlen(key) * sizeof(char))+1;

    hipMalloc( (void**)&dev_key, textSize);
    hipMalloc( (void**)&dev_text, textSize);

    hipMemcpy( dev_key, key, keySize, hipMemcpyHostToDevice);
    hipMemcpy( dev_text, text, textSize, hipMemcpyHostToDevice);

    //printf("%i %i %i\n", (int)sizeof(text), (strlen(text)+1)*sizeof(char), (strlen(text))*sizeof(char));

    printf("Key: %s(%d)\nText: '%s'(%d)\n", key, keySize, text,textSize);
    caesarCipher<<<textSize, 1>>>(dev_key, dev_text, textSize, keySize);


    char * result = (char *)malloc(textSize);
    hipMemcpy(result, dev_text, textSize, hipMemcpyDeviceToHost);

    printf("Output:");
    printf(" '%s'\n", result);
    printf("Length: %i", strlen(result));

    unCaesarCipher<<<textSize, 1>>>(dev_key, dev_text, textSize, keySize);

    hipMemcpy(result, dev_text, textSize, hipMemcpyDeviceToHost);

    printf("Output:");
    printf(" '%s'\n", result);
    printf("Length: %i", strlen(result));


    printf("Clearing Memory...\n");

    hipFree(dev_text);
    hipFree(dev_key);
    free(result);


    return 0;
}

__global__ void caesarCipher(char *key, char *text, int tlength, int klength)
{
    int tid = blockIdx.x;
    if (tid < tlength)
    {
        //printf("%i says - %s\n", tid, text);
        char t = text[tid];
        text[tid] = ((int)text[tid] + (int)key[tid % klength])%127;
        printf("%c -> %c - %d\n",t,text[tid], tid);
    }

}

__global__ void unCaesarCipher(char *key, char *text, int tlength, int klength)
{
    int tid = blockIdx.x;
    if (tid < tlength)
    {
        //printf("%i says - %s\n", tid, text);
        char t = text[tid];
        text[tid] = ((int)text[tid] - (int)key[tid % klength])%127;
        printf("%c -> %c - %d\n",t,text[tid], tid);
    }

}

void getInfo( void )
{
    hipDeviceProp_t p;

    if(getDevicePresent())
    {
        hipGetDeviceProperties( &p, 0);
        printf(" -- Information & Properties about CUDA device 0 -- \n\n");
        printf("\tCompute Capability: %i.%i\n", p.major, p.minor);
        printf("\tDevice Name: %s\n", p.name);
        printf("\tClock Rate: %d\n", p.clockRate);
        printf("\tGlobal Memory: %dMiB\n", p.totalGlobalMem/(1024*1024));
        printf("\n -- End of Information -- \n");
    }else
    {
        printf(" -- Warning: No CUDA Device Detected :'( -- \n");
        printf(" -- This software might not operate as   -- \n -- Expected.                            -- \n");
    }
}

bool getDevicePresent( void )
{
    hipDeviceProp_t p;
    hipGetDeviceProperties( &p, 0);
    return (p.major != 0);
}
